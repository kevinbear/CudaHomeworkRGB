#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include "bmp.h"
__global__ void GPU_ChangeRGB(int R[MaxBMPSizeX][MaxBMPSizeY], int G[MaxBMPSizeX][MaxBMPSizeY], int B[MaxBMPSizeX][MaxBMPSizeY], int *GPU_R, int *GPU_G, int *GPU_B, int rgb_offest[3], int width, int height){
	
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(rgb_offest[0] == 0 && rgb_offest[1] == 0){
		GPU_R[index] = R[blockIdx.x][threadIdx.x];
		GPU_G[index] = G[blockIdx.x][threadIdx.x];
		GPU_B[index] = B[blockIdx.x][threadIdx.x] + rgb_offest[2];
	}
	else if (rgb_offest[1] == 0 && rgb_offest[2] == 0){
		GPU_R[index] = R[blockIdx.x][threadIdx.x] + rgb_offest[0];
		GPU_G[index] = G[blockIdx.x][threadIdx.x];
		GPU_B[index] = B[blockIdx.x][threadIdx.x];
	}
	else if (rgb_offest[0] == 0 && rgb_offest[2] == 0){
		GPU_R[index] = R[blockIdx.x][threadIdx.x];
		GPU_G[index] = G[blockIdx.x][threadIdx.x] + rgb_offest[1];
		GPU_B[index] = B[blockIdx.x][threadIdx.x];
	}
	else{
		GPU_R[index] = R[blockIdx.x][threadIdx.x] + rgb_offest[0];
		GPU_G[index] = G[blockIdx.x][threadIdx.x] + rgb_offest[1];
		GPU_B[index] = B[blockIdx.x][threadIdx.x] + rgb_offest[2];
	}
}

void Cuda_Change_RGB(int (&R)[MaxBMPSizeX][MaxBMPSizeY], int (&G)[MaxBMPSizeX][MaxBMPSizeY], int (&B)[MaxBMPSizeX][MaxBMPSizeY],
                     int (&r)[MaxBMPSizeX][MaxBMPSizeY], int (&g)[MaxBMPSizeX][MaxBMPSizeY], int (&b)[MaxBMPSizeX][MaxBMPSizeY],
                     int width,int height,int (&rgb_offest)[3]){
		
	int *GPU_R, *GPU_G, *GPU_B;
	size_t pitch_r,pitch_g,pitch_b;
	//allocated gpu memory to 3 2-D array (R,G,B)
	hipMallocPitch(&GPU_R, &pitch_r, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY);
	hipMallocPitch(&GPU_G, &pitch_g, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY);
	hipMallocPitch(&GPU_B, &pitch_b, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY);
	// from cpu replicate data to gpu memory	
	hipMemcpy2D(GPU_R, pitch_r, R, sizeof(int*)*MaxBMPSizeX, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyHostToDevice);
	hipMemcpy2D(GPU_G, pitch_g, G, sizeof(int*)*MaxBMPSizeX, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyHostToDevice);
	hipMemcpy2D(GPU_B, pitch_b, B, sizeof(int*)*MaxBMPSizeX, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyHostToDevice);
	// kernel function
	GPU_ChangeRGB<<<height,width>>>(R, G, B, GPU_R, GPU_G, GPU_B, rgb_offest, width, height);
	// from GPU copy processed data back to CPU memroy
	hipMemcpy2D(r, sizeof(int*)*MaxBMPSizeX, GPU_R, pitch_r, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyDeviceToHost);
	hipMemcpy2D(g, sizeof(int*)*MaxBMPSizeX, GPU_G, pitch_g, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyDeviceToHost);
	hipMemcpy2D(b, sizeof(int*)*MaxBMPSizeX, GPU_B, pitch_b, sizeof(int*)*MaxBMPSizeX, MaxBMPSizeY, hipMemcpyDeviceToHost);
}
