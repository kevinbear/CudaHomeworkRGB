#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include "bmp.h"
__global__ void GPU_ChangeRGB(int *D_R, int *D_G, int *D_B, int *GPU_R, int *GPU_G, int *GPU_B, int *offest){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	printf("r:%d g:%d b:%d\n",offest[0],offest[1],offest[2]);
	if(offest[0] == 0 && offest[1] == 0){
		GPU_R[index] = D_R[index];
		GPU_G[index] = D_G[index];
		GPU_B[index] = D_B[index] + offest[2];
	}
	else if (offest[1] == 0 && offest[2] == 0){
		GPU_R[index] = D_R[index] + offest[0];
		GPU_G[index] = D_G[index];
		GPU_B[index] = D_B[index];
	}
	else if (offest[0] == 0 && offest[2] == 0){
		GPU_R[index] = D_R[index];
		GPU_G[index] = D_G[index] + offest[1];
		GPU_B[index] = D_B[index];
	}
	else{
		GPU_R[index] = D_R[index] + offest[0];
		GPU_G[index] = D_G[index] + offest[1];
		GPU_B[index] = D_B[index] + offest[2];
	}
}

void Cuda_Change_RGB(int (&R)[MaxBMPSizeX][MaxBMPSizeY], int (&G)[MaxBMPSizeX][MaxBMPSizeY], int (&B)[MaxBMPSizeX][MaxBMPSizeY],
                     int (&r)[MaxBMPSizeX][MaxBMPSizeY], int (&g)[MaxBMPSizeX][MaxBMPSizeY], int (&b)[MaxBMPSizeX][MaxBMPSizeY],
                     int width,int height,int (&rgb_offest)[3]){
	int i,j;
	int Flat_R[width*height];
	int Flat_G[width*height];
	int Flat_B[width*height];
	// 2-D converting to 1-D array
	for(i=0;i<height;i++){
		for(j=0;j<width;j++){
			Flat_R[i*width+j] = R[i][j];
			Flat_G[i*width+j] = G[i][j];
			Flat_B[i*width+j] = B[i][j];
		}	
	}
	int *GPU_R = NULL, *GPU_G = NULL, *GPU_B = NULL,*D_R = NULL,*D_G = NULL,*D_B = NULL;
	int *offest = NULL;	
	//allocated gpu memory to 3 2-D array (R,G,B)
	hipMalloc(&D_R, sizeof(int)*width*height);
	hipMalloc(&D_G, sizeof(int)*width*height);
	hipMalloc(&D_B, sizeof(int)*width*height);
	hipMalloc(&GPU_R, sizeof(int)*width*height);
	hipMalloc(&GPU_G, sizeof(int)*width*height);
	hipMalloc(&GPU_B, sizeof(int)*width*height);
	hipMalloc(&offest, sizeof(int)*3);	
	hipMemset(D_R,255,sizeof(int)*width*height);
	hipMemset(D_G,255,sizeof(int)*width*height);
	hipMemset(D_B,255,sizeof(int)*width*height);
	hipMemset(GPU_R,255,sizeof(int)*width*height);
	hipMemset(GPU_G,255,sizeof(int)*width*height);
	hipMemset(GPU_B,255,sizeof(int)*width*height);
	hipMemset(offest,0,sizeof(int)*3);
	// from cpu replicate data to gpu memory	
	hipMemcpy(D_R, Flat_R, sizeof(int)*width*height, hipMemcpyHostToDevice);
	hipMemcpy(D_G, Flat_G, sizeof(int)*width*height, hipMemcpyHostToDevice);
	hipMemcpy(D_B, Flat_B, sizeof(int)*width*height, hipMemcpyHostToDevice);
	hipMemcpy(offest,rgb_offest,sizeof(int)*3, hipMemcpyHostToDevice);
	// kernel function
	GPU_ChangeRGB<<<height,width>>>(D_R, D_G, D_B, GPU_R, GPU_G, GPU_B, offest);
	// from GPU copy processed data back to CPU memroy
	int Pass_R[width*height];
	int Pass_G[width*height];
	int Pass_B[width*height];	
	hipMemcpy(Pass_R, GPU_R, sizeof(int)*width*height, hipMemcpyDeviceToHost);
	hipMemcpy(Pass_G, GPU_G, sizeof(int)*width*height, hipMemcpyDeviceToHost);
	hipMemcpy(Pass_B, GPU_B, sizeof(int)*width*height, hipMemcpyDeviceToHost);
	// 1-D converting to 2-D array
	for(i=0;i<height;i++){
		for(j=0;j<width;j++){
			r[i][j] = Pass_R[i*width+j];
			g[i][j] = Pass_G[i*width+j];
			b[i][j] = Pass_B[i*width+j];
		}	
	}
}
